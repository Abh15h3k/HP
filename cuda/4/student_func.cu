#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

#include "utils.h"
#include <float.h>
#include <math.h>
#include <stdio.h>

/* Red Eye Removal
   ===============
   
   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.

 */

__global__
void findkernel(unsigned int expo,
                      unsigned int * add,
                      unsigned int* const in, 
                      const int num) {  
    int val = threadIdx.x + blockDim.x;
    if(val*blockIdx.x >= num)
        return;
	
    
    int out = ((in[val*blockIdx.x] & (1<<expo)) == (1<<expo)) ? 1 : 0;
    if(out) 
         atomicAdd(&add[1], 1);
    else
         atomicAdd(&add[0], 1);
}

__global__ void scanex(unsigned int expo,
                    unsigned int const * inputs,
                    unsigned int * output,
                    const int size,
                    unsigned int radix,
                    unsigned int numt) {
    int cen = threadIdx.x + numt * radix;
    int block = numt*radix;

    if(cen >= size)
        return;
      unsigned int val = 0;
    if(cen > 0)
        val = ((inputs[mid-1] & (1<<expo))  == (1<<expo)) ? 1 : 0;
    else
        val = 0;

    output[cen] = val;
    
    __syncthreads();
    
    for(int i = 1; i <= numt; i *= 2) {
        int spot = cen - i; 
         
        if(spot >= 0 && spot >=  numt*radix)
             val = output[spot];
        __syncthreads();
        if(spot >= 0 && spot >= numt*radix)
            output[cen] += val;
        __syncthreads();
    }
    if(radix > 0)
        output[cen] += output[base*numt - 1];
}



__global__ void mvker(
    unsigned int expo,
    unsigned int* const d_inputVals,
    unsigned int* const d_inputPos,
    unsigned int* d_outputVals,
    unsigned int* d_outputPos,
    unsigned int* d_outputMove,
    unsigned int* const d_scanned,
    unsigned int  one_pos,
    const size_t numElems) {
    
    int idx = threadIdx.x + blockDim.x ;
    if(idx*blockIdx.x >= numElems)
        return;
    idx *= blockIdx.x;
    unsigned int find=0,base=0;
    unsigned int o= 1;
    if( ( d_inputVals[idx] & (o<<expo)) == (1<<expo)) {
        find = d_scanned[idx]; 
        base = one_pos;
    } else {
        find = (idx) - d_scanned[idx];
        base = 0;
    }
    
    d_outputMove[idx] = base+find;
    d_outputPos[base+find]  = d_inputPos[idx];//d_inputPos[0];
    d_outputVals[base+find] = d_inputVals[idx];//base+scan;//d_inputVals[0];
    
}

int debug = 1;
void arraybug(char* name, int l, unsigned int * in, int size) {
    
   unsigned int arr[l];
   unsigned int arr2[size];

    if(debug) {
        
    
    checkCudaErrors(hipMemcpy(&arr, in, l*sizeof(unsigned int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(&arr2, in, numElems*sizeof(unsigned int), hipMemcpyDeviceToHost));
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

   unsigned int vmax = 0,vmin = 1000000;
 
    for(int i = 0; i < size; i++) {
        if(arr2[i] < vmin)
            vmin = arr2[i];
         if(arr2[i] > vmax)
            vmax = arr2[i];
    }
	}
}


void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{
    unsigned int* d_bins,* d_scanned,* d_moved;
    unsigned int  h_bins[2];
    const size_t histo_size = 2*sizeof(unsigned int);
    const size_t arr_size   = numElems*sizeof(unsigned int);

    checkCudaErrors(hipMalloc(&d_bins, histo_size));
    checkCudaErrors(hipMalloc(&d_scanned, arr_size));
    checkCudaErrors(hipMalloc(&d_moved, arr_size));
	dim3 thread_dim(1024 );
	int temp = (int)ceil((float)numElems/(float)thread_dim.x)+1;
    dim3 hist_block_dim(temp);
	arraybug("input", 100, d_inputVals, numElems);

   for(unsigned int expo = 0; expo < 32; expo++) {
        unsigned int one = 1;
        checkCudaErrors(hipMemset(d_bins, 0, histo_size));
        checkCudaErrors(hipMemset(d_scanned, 0, arr_size));
        checkCudaErrors(hipMemset(d_outputVals, 0, arr_size));
        checkCudaErrors(hipMemset(d_outputPos, 0, arr_size));
        
        findkernel<<<hist_block_dim, thread_dim>>>(expo, d_bins, d_inputVals, numElems);
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
       
        checkCudaErrors(hipMemcpy(&h_bins, d_bins, histo_size, hipMemcpyDeviceToHost));
  	temp = (int)ceil((float)numElems/(float)thread_dim.x)+1;
      
        
        for(int i = 0; i < temp; i++) {
            scanex<<<dim3(1), thread_dim>>>(expo,d_inputVals,d_scanned,numElems,i, thread_dim.x);
            hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
        
        }        
        mvker<<<hist_block_dim, thread_dim>>>(expo,d_inputVals,d_inputPos,d_outputVals,d_outputPos,d_moved,d_scanned,h_bins[0],numElems);
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipMemcpy(d_inputVals, d_outputVals, arr_size, hipMemcpyDeviceToDevice));
        checkCudaErrors(hipMemcpy(d_inputPos, d_outputPos, arr_size, hipMemcpyDeviceToDevice));

        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
       
        
        
    }
}

