#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Dynamic Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/

#include<limits.h>
#include<float.h>
#include<math.h>
#include<stdio.h>
#include ""
#include "utils.h"


__global__ void reduction((float* d_out, const float * d_in, int size,bool flag)
{
	extern __shared__ float data[];
	
	int x = thread.x;
	int gid = blockDim.x*blockIdx.x + x;
	
	if(gid>=size) {
		data[x] =d_in[0];	
	}
	else
		data[x] = d_in[gid];
	__syncthreads();
	for(int i = blockDim.x/2, i>0; i = i/2)
	{
		if(x < i)
		{
			if(flag)
			{
				float mini;
				if(data[x]<data[x+i])
					mini = data[x];
				else
					mini = data[x+i];
				data[x] = mini;
			}
			else
			{
				float mini;
				if(data[x]>data[x+i])
					mini = data[x];
				else
					mini = data[x+i];
				data[x] = mini;

			}
		}
		__syncthreads();
	}
	if(x == 0) {
		d_out[blockIdx.x] = data[0];
	}
}

__global__ void atomic(unsigned int* out_histo,  const float * d_in, int num, int size, float minal, float dim) {
	int x = threadIdx.x;
	int gid = x + blockDim.x*blockIdx.x;
	if (gid >= size) return;
	int flg  = ((d_in[gid] - minal)*num) / dim;		if(flg == num)
	{
		flg = num -1;
	}
	atomicAdd(&(out_histo[flg]), 1);
}

__global__ void find(unsigned int* d_out,const unsigned int* d_in, int size) {
	extern __shared__ unsigned int temp[];
	int x = threadIdx.x;
	int px = 0,py=1;
	temp[x] = x>0? d_in[x-1]:0; //exclusive scan
	__syncthreads();

	//double buffered
	for (int i = 1; i < size; i = i /2) {
		px = 1 - px;
		py = 1 - px;
		if (x >= i) temp[size*px + x] = temp[size*py + x]+temp[size*py + x - off];
		else temp[size*px + tid] = temp[size*py + x];
		__syncthreads();
	}
	d_out[x] = temp[py*size + x];
}

float convert(const float* const logd int isize,bool isMin) {
	int t = BLOCK_SIZE;
	float* d_current_in = NULL;
	int size = isize;
	int blocks = ceil(1.0f*size / t); 
	while (true) {
		float* d_out;
		checkCudaErrors(hipMalloc(&d_out, blocks * sizeof(float)));
		//call reduce kernel: if first iteration use original vector, otherwise use the last intermediate result.
		if (d_current_in == NULL) reduction << <blocks, t, threads * sizeof(float) >> > (d_out, logd, size, isMin);
		else reduction << <blocks, t, threads * sizeof(float) >> > (d_out, d_current_in, size, isMin);;
		hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

		//free last intermediate result
		if (d_current_in != NULL) checkCudaErrors(hipFree(d_current_in));

		if (blocks == 1) {
			//end of reduction reached
			float h_out;
			checkCudaErrors(hipMemcpy(&h_out, d_out, sizeof(float), hipMemcpyDeviceToHost));
			return h_out;
		}
		size = blocks;
		blocks = ceil(1.0f*size / t); 
		if (blocks == 0)blocks++;
		d_current_in = d_out;//point to new intermediate result
		
	}
	
}

unsigned int* calhis(const float* const logd, int nbin, int size, float vmin, float vrng) {
	unsigned int* error;
	checkCudaErrors(hipMalloc(&error, numBins * sizeof(unsigned int)));
	checkCudaErrors(hipMemset(error, 0, numBins * sizeof(unsigned int)));
	int threads = BLOCK_SIZE;
	int blocks = ceil(1.0f*size / threads);
	atomic << <blocks, threads >> >(error, logd, nbin,size, vmin, vrng);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
	return error;
}
void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */

	int size = numCols * numRows;
	max_logLum = convert(d_logLuminance,size,false);
	min_logLum = convert(d_logLuminance,size,true);
	
	unsigned int *out = compute(d_logLuminance, numBins, input_size, min_logLum, range);
	
	unsigned int out2 = calhis(d_logLuminance, numBins, input_size, min_logLum, range);
	find<<<1,numBins,2*numBins*sizeof(unsigned int)>>>(d_cdf,d_gisto, numBins);
}
